
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
//__global__ int *arrayLocal_16, *arrayLocal_4;

__device__ void T2x2H(int *iCoeff, int valRound)
{
    int valT1, valT2;
    iCoeff[0] += iCoeff[3];
    iCoeff[1] -= iCoeff[2];
    valT1 = ((iCoeff[0] - iCoeff[1] + valRound) >> 1);
    valT2 = iCoeff[2];
    iCoeff[2] = valT1 - iCoeff[3];
    iCoeff[3] = valT1 - valT2;
    iCoeff[0] -= iCoeff[3];
    iCoeff[1] += iCoeff[2];
}

__device__ void TOdd(int *iCoeff, int dummy)
{
    iCoeff[1] -= iCoeff[2];
    iCoeff[0] += iCoeff[3];
    iCoeff[2] += ((iCoeff[1] + 1) >> 1);
    iCoeff[3] = ((iCoeff[0] + 1) >> 1) - iCoeff[3];
    iCoeff[1] -= ((3* iCoeff[0] + 4) >> 3);
    iCoeff[0] += ((3* iCoeff[1] + 4) >> 3);
    iCoeff[3] -= ((3* iCoeff[2] + 4) >> 3);
    iCoeff[2] += ((3* iCoeff[3] + 4) >> 3);
    iCoeff[3] += (iCoeff[1] >> 1);
    iCoeff[2] -= ((iCoeff[0] + 1) >> 1);
    iCoeff[1] -= iCoeff[3];
    iCoeff[0] += iCoeff[2];
}

__device__ void TOddOdd(int *iCoeff, int dummy)
{
    int valT1, valT2;
    iCoeff[1] = -iCoeff[1];
    iCoeff[2] = -iCoeff[2];
    iCoeff[3] += iCoeff[0];
    iCoeff[2] -= iCoeff[1];
    iCoeff[0] -= (valT1 = iCoeff[3] >> 1);
    iCoeff[1] += (valT2 = iCoeff[2] >> 1);
    iCoeff[0] += ((iCoeff[1] * 3 + 4) >> 3);
    iCoeff[1] -= ((iCoeff[0] * 3 + 3) >> 2);
    iCoeff[0] += ((iCoeff[1] * 3 + 3) >> 3);
    iCoeff[1] -= valT2;
    iCoeff[0] += valT1;
    iCoeff[2] += iCoeff[1];
    iCoeff[3] -= iCoeff[0];
}

__device__ void FwdPermute(int *arrayInput)
{
    int i;
    int arrayTemp[16];
    int FwdPermArr[16] = {
        0, 8, 4, 6, 2, 10, 14, 12,
        1, 11, 15, 13, 9, 3, 7, 5
    };
    for (i = 0; i <= 15; i++)
        arrayTemp[FwdPermArr[i]] = arrayInput[i];
    for (i = 0; i <= 15; i++)
        arrayInput[i] = arrayTemp[i];
}

__device__ void (*pointerFunct_FCT[8]) (int *arg1, int arg2) = {
    T2x2H, T2x2H, T2x2H, T2x2H,
    T2x2H, TOdd, TOdd, TOddOdd
};
__device__ void FCT4x4(int *iCoeff)
{
    int arrayLocal[4];
    int arrayTemp[8][4] = {
        { 0, 3, 12, 15},
        { 5, 6, 9, 10},
        { 1, 2, 13, 14},
        { 4, 7, 8, 11},
        { 0, 1, 4, 5},
        { 2, 3, 6, 7},
        { 8, 12, 9, 13},
        { 10, 11, 14, 15}
    };


    int arg2Array[8] = { 0, 0, 0, 0, 1, 0, 0, 0};
    int i, j;

    for(i = 0; i < 8; i++)
    {
        for(j = 0; j < 4; j++)
            arrayLocal[j] = iCoeff[arrayTemp[i][j]];

        (*pointerFunct_FCT[i]) (arrayLocal, arg2Array[i]);

        for(j = 0; j < 4; j++)
            iCoeff[arrayTemp[i][j]] = arrayLocal[j];
    }

    FwdPermute(iCoeff);
}

__device__ void T2x2hEnc(int *iCoeff, int dummy)
{
    int valT1, valT2;
    iCoeff[0] += iCoeff[3];
    iCoeff[1] -= iCoeff[2];
    valT1 = iCoeff[3];
    valT2 = iCoeff[2];
    iCoeff[2] = ((iCoeff[0] - iCoeff[1]) >> 1) - valT1;
    iCoeff[3] = valT2 + (iCoeff[1] >> 1);
    iCoeff[1] += iCoeff[2];
    iCoeff[0] -= (iCoeff[3] * 3 + 4) >> 3;
}

__device__ void FwdRotate(int *iCoeff, int dummy)
{
    iCoeff[1] -= (iCoeff[0] + 1) >> 1;
    iCoeff[0] += (iCoeff[1] + 1) >> 1;
}

__device__ void FwdScale(int *iCoeff, int dummy)
{
    iCoeff[1] -= (iCoeff[0] * 3 + 0) >> 4;
    iCoeff[1] -= (iCoeff[0] >> 7);
    iCoeff[1] += (iCoeff[0] >> 10);
    iCoeff[0] -= (iCoeff[1] * 3 + 0) >> 3;
    iCoeff[1] = (iCoeff[0] >> 1) - iCoeff[1];
    iCoeff[0] -= iCoeff[1];
}

__device__ void FwdTOddOdd(int *iCoeff, int dummy)
{
    int valT1, valT2;
    iCoeff[3] += iCoeff[0];
    iCoeff[2] -= iCoeff[1];
    valT1 = iCoeff[3] >> 1;
    valT2 = iCoeff[2] >> 1;
    iCoeff[0] -= valT1;
    iCoeff[1] += valT2;
    iCoeff[0] += (iCoeff[1] * 3 + 4) >>3;
    iCoeff[1] -= (iCoeff[0] * 3 + 2) >> 2;
    iCoeff[0] += (iCoeff[1] * 3 + 6) >> 3;
    iCoeff[1] -= valT2;
    iCoeff[0] += valT1;
    iCoeff[2] += iCoeff[1];
    iCoeff[3] -= iCoeff[0];
}

__device__ void (*pointerFunct[17]) (int *arg1, int arg2) = {
    T2x2hEnc, T2x2hEnc, T2x2hEnc, T2x2hEnc,
    FwdScale, FwdScale, FwdScale, FwdScale,
    FwdRotate, FwdRotate, FwdRotate, FwdRotate,
    FwdTOddOdd, T2x2H, T2x2H, T2x2H, T2x2H
};

__device__ void OverlapPreFilter4x4(int *iCoeff)
{
    int arrayLocal[4];
    int arrayTemp[17][4] = {
        { 0, 3, 12, 15},
        { 1, 2, 13, 14},
        { 4, 7, 8, 11},
        { 5, 6, 9, 10},

        { 0, 15, -1, -1},
        { 1, 14, -1, -1},
        { 4, 11, -1, -1},
        { 5, 10, -1, -1},

        { 13, 12, -1, -1},
        { 9, 8, -1, -1},
        { 7, 3, -1, -1},
        { 6, 2, -1, -1},

        { 10, 11, 14, 15},
        { 0, 12, 3, 15},
        { 1, 2, 13, 14},
        { 4, 7, 8, 11},
        { 5, 6, 9, 10}
    };


    int i, j;

    for(i = 0; i < 17; i++)
    {
        for(j = 0; j < 4; j++)
            if(arrayTemp[i][j] >= 0)
                arrayLocal[j] = iCoeff[arrayTemp[i][j]];

        (*pointerFunct[i]) (arrayLocal, 0);

        for(j = 0; j < 4; j++)
            if(arrayTemp[i][j] >= 0)
                iCoeff[arrayTemp[i][j]] = arrayLocal[j];
    }
}

__device__ void OverlapPreFilter4(int *iCoeff)
{
    int arrayLocal[2];
    iCoeff[0] += iCoeff[3];
    iCoeff[1] += iCoeff[2];
    iCoeff[3] -= ((iCoeff[0] + 1) >> 1);
    iCoeff[2] -= ((iCoeff[1] + 1) >> 1);
    arrayLocal[0] = iCoeff[2], arrayLocal[1] = iCoeff[3];
    FwdRotate(arrayLocal, 0);
    iCoeff[2] = arrayLocal[0], iCoeff[3] = arrayLocal[1];
    iCoeff[3] = -iCoeff[3];
    iCoeff[2] = -iCoeff[2];
    iCoeff[0] -= iCoeff[3];
    iCoeff[1] -= iCoeff[2];
    iCoeff[3] += ( iCoeff[0] >> 1);
    iCoeff[2] += ( iCoeff[1] >> 1);
    iCoeff[0] -= ((iCoeff[3] * 3 + 4) >> 3);
    iCoeff[1] -= ((iCoeff[2] * 3 + 4) >> 3);
    arrayLocal[0] = iCoeff[0], arrayLocal[1] = iCoeff[3];
    FwdScale(arrayLocal, 0);
    iCoeff[0] = arrayLocal[0], iCoeff[3] = arrayLocal[1];

    arrayLocal[0] = iCoeff[1], arrayLocal[1] = iCoeff[2];
    FwdScale(arrayLocal, 0);
    iCoeff[1] = arrayLocal[0], iCoeff[2] = arrayLocal[1];
    iCoeff[3] += ((iCoeff[0] + 1) >> 1);
    iCoeff[2] += ((iCoeff[1] + 1) >> 1);
    iCoeff[0] -= iCoeff[3];
    iCoeff[1] -= iCoeff[2];
}

__device__ void OverlapPreFilter2x2(int *iCoeff)
{
    iCoeff[0] += iCoeff[3];
    iCoeff[1] += iCoeff[2];
    iCoeff[3] -= ((iCoeff[0] + 1) >> 1);
    iCoeff[2] -= ((iCoeff[1] + 1) >> 1);
    iCoeff[1] -= ((iCoeff[0] + 2) >> 2);
    iCoeff[0] -= (iCoeff[1] >> 5);
    iCoeff[0] -= (iCoeff[1] >> 9);
    iCoeff[0] -= (iCoeff[1] >> 13);
    iCoeff[0] -= ((iCoeff[1] + 1) >> 1);
    iCoeff[1] -= ((iCoeff[0] + 2) >> 2);
    iCoeff[3] += ((iCoeff[0] + 1) >> 1);
    iCoeff[2] += ((iCoeff[1] + 1) >> 1);
    iCoeff[0] -= iCoeff[3];
    iCoeff[1] -= iCoeff[2];
}

__device__ void OverlapPreFilter2(int *iCoeff)
{
    iCoeff[1] -= ((iCoeff[0] + 2) >> 2);
    iCoeff[0] -= (iCoeff[1] >> 13);
    iCoeff[0] -= (iCoeff[1] >> 9);
    iCoeff[0] -= (iCoeff[1] >> 5);
    iCoeff[0] -= ((iCoeff[1] + 1) >> 1);
    iCoeff[1] -= ((iCoeff[0] + 2) >> 2);
}

__global__ void EncFirstStagePreFiltering(int *image, int numRows, int numCols)
{
    int i, j;
    int arrayLocal[16];
    int block_i = threadIdx.x, block_j = threadIdx.y;
    int macro_i = blockIdx.x, macro_j = blockIdx.y;

    for (i = 0; i < 4; ++i)
    {
        for (j = 0; j < 4; ++j)
        {
            arrayLocal[i*4 + j] = image[ (macro_i*16 + block_i*4 + i) * numCols + macro_j*16 + block_j*4 + j];
        }
    }

    FCT4x4(arrayLocal);

    for (i = 0; i < 4; ++i)
    {
        for (j = 0; j < 4; ++j)
        {
            image[(macro_i*16 + block_i*4 + i) * numCols + macro_j*16 + block_j*4 + j] = arrayLocal[i*4 + j];
        }
    }
}

__global__ void EncSecondStagePreFiltering(int* image, int numRows, int numCols)
{
    int i, j;
    int arrayLocal[16];
    int macro_i = blockIdx.x, macro_j = blockIdx.y;

    for (i = 0; i < 4; ++i)
    {
        for (j = 0; j < 4; ++j)
        {
            arrayLocal[i*4 + j] = image[ (macro_i*16 + i*4) * numCols + macro_j*16 + j*4];
        }
    }
    FCT4x4(arrayLocal);
    for (i = 0; i < 4; ++i)
    {
        for (j = 0; j < 4; ++j)
        {
            image[(macro_i*16 + i*4) * numCols + macro_j*16 + j*4] = arrayLocal[i*4 + j];
        }
    }
}

__global__ void EncFirstStageOverlapFilter(int* image, int numRows, int numCols)
{
    int arrayLocal_16[16], arrayLocal_4[4];
    int block_i = blockIdx.x, block_j = blockIdx.y, i, j;

    //numRows /= 4;
    //numCols /= 4;

    // 4x4 blocks
    for( i = 0; i < 4; i++)
        for( j = 0; j < 4; j++)
            arrayLocal_16[i*4+j] = image[((block_i*4 + i + 2)*4)*numCols + (block_j*4 + j + 2)*4];

    OverlapPreFilter4x4(arrayLocal_16);
    for( i = 0; i < 4; i++)
        for( j = 0; j < 4; j++)
            image[((block_i*4 + i + 2)*4)*numCols + (block_j*4 + j + 2)*4] = arrayLocal_16[i*4+j];
    //4x4 block end

    if(block_j == 0)
    {
        //left edge
        for(i = 0; i < 2; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[((block_i*4+j+2)*4)*numCols + i*4];
            OverlapPreFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[((block_i*4+j+2)*4)*numCols + i*4] = arrayLocal_4[j];
        }
        // right edge
        for(i = numCols/4-2; i < numCols/4; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[((block_i*4+j+2)*4)*numCols + i*4];
            OverlapPreFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[((block_i*4+j+2)*4)*numCols + i*4] = arrayLocal_4[j];
        }
    }


    if(block_i == 0)
    {
        //top edge
        for(i = 0; i < 2; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[(i*4)*numCols + (block_j*4+j+2)*4];
            OverlapPreFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[(i*4)*numCols + (block_j*4+j+2)*4] = arrayLocal_4[j];
        }
        //bottom edge
        for(i = numRows/4-2; i < numRows/4; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[(i*4)*numCols + (block_j*4+2+j)*4];
            OverlapPreFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[(i*4)*numCols + (block_j*4+j+2)*4] = arrayLocal_4[j];
        }
    }

    if(block_j == 0 && block_i == 0)
    {
        // top left
        arrayLocal_4[0] = image[(0)*numCols + 0], arrayLocal_4[1] = image[(0)*numCols + 4];
        arrayLocal_4[2] = image[(4)*numCols + 0], arrayLocal_4[3] = image[(4)*numCols + 4];
        OverlapPreFilter4(arrayLocal_4);
        image[(0)*numCols + 0] = arrayLocal_4[0], image[(0)*numCols + 4] = arrayLocal_4[1];
        image[(4)*numCols + 0] = arrayLocal_4[2], image[(4)*numCols + 4] = arrayLocal_4[3];

        // top right
        arrayLocal_4[0] = image[(0)*numCols + (numCols/4-2)*4], arrayLocal_4[1] = image[(0)*numCols + (numCols/4-1)*4];
        arrayLocal_4[2] = image[(4)*numCols + (numCols/4-2)*4], arrayLocal_4[3] = image[(4)*numCols + (numCols/4-1)*4];
        OverlapPreFilter4(arrayLocal_4);
        image[(0)*numCols + (numCols/4-2)*4] = arrayLocal_4[0], image[(0)*numCols + (numCols/4-1)*4] = arrayLocal_4[1];
        image[(4)*numCols + (numCols/4-2)*4] = arrayLocal_4[2], image[(4)*numCols + (numCols/4-1)*4] = arrayLocal_4[3];

        // bottom left
        arrayLocal_4[0] = image[((numRows/4-2)*4)*numCols + 0], arrayLocal_4[1] = image[((numRows/4-2)*4)*numCols + 4];
        arrayLocal_4[2] = image[((numRows/4-1)*4)*numCols + 0], arrayLocal_4[3] = image[((numRows/4-1)*4)*numCols + 4];
        OverlapPreFilter4(arrayLocal_4);
        image[((numRows/4-2)*4)*numCols + 0] = arrayLocal_4[0], image[((numRows/4-2)*4)*numCols + 4] = arrayLocal_4[1];
        image[((numRows/4-1)*4)*numCols + 0] = arrayLocal_4[2], image[((numRows/4-1)*4)*numCols + 4] = arrayLocal_4[3];

        // bottom right
        arrayLocal_4[0] = image[((numRows/4-2)*4)*numCols + (numCols/4-2)*4], arrayLocal_4[1] = image[((numRows/4-2)*4)*numCols + (numCols/4-1)*4];
        arrayLocal_4[2] = image[((numRows/4-1)*4)*numCols + (numCols/4-2)*4], arrayLocal_4[3] = image[((numRows/4-1)*4)*numCols + (numCols/4-1)*4];
        OverlapPreFilter4(arrayLocal_4);
        image[((numRows/4-2)*4)*numCols + (numCols/4-2)*4] = arrayLocal_4[0], image[((numRows/4-2)*4)*numCols + (numCols/4-1)*4] = arrayLocal_4[1];
        image[((numRows/4-1)*4)*numCols + (numCols/4-2)*4] = arrayLocal_4[2], image[((numRows/4-1)*4)*numCols + (numCols/4-1)*4] = arrayLocal_4[3];
    }
}

__global__ void EncSecondStageOverlapFilter(int* image, int numRows, int numCols)
{
    int arrayLocal_16[16], arrayLocal_4[4];
    int block_i = blockIdx.x, block_j = blockIdx.y, i, j;

    // 4x4 blocks
    for( i = 0; i < 4; i++)
    {
        for( j = 0; j < 4; j++)
            arrayLocal_16[i*4+j] = image[ (block_i*4 + 2 + i)*numCols + block_j*4 + 2 + j];
    }
    OverlapPreFilter4x4(arrayLocal_16);
    for( i = 0; i < 4; i++)
    {
        for( j = 0; j < 4; j++)
            image[(block_i*4 + 2 + i)*numCols + block_j*4 + 2 + j] = arrayLocal_16[i*4+j];
    }
    //4x4 block end

    if(block_j == 0)
    {
        //left edge
        for(i = 0; i < 2; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[(block_i*4 + 2+j)*numCols + i];
            OverlapPreFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[(block_i*4 + 2+j)*numCols + i] = arrayLocal_4[j];
        }
        // right edge
        for(i = numCols-2; i < numCols; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[(block_i*4 + 2 + j)*numCols + i];
            OverlapPreFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[(block_i*4 + 2+j)*numCols + i] = arrayLocal_4[j];
        }
    }


    if(block_i == 0)
    {
        //top edge
        for(i = 0; i < 2; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[(i)*numCols + block_j*4 + 2+j];
            OverlapPreFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[(i)*numCols + block_j*4 + 2+j] = arrayLocal_4[j];
        }
        //bottom edge
        for(i = numRows-2; i < numRows; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[(i)*numCols + block_j*4 + 2 + j];
            OverlapPreFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[(i)*numCols + block_j*4 + 2 + j] = arrayLocal_4[j];
        }
    }

    if(block_i == 0 && block_j == 0)
    {
        // top left
        arrayLocal_4[0] = image[(0)*numCols + 0], arrayLocal_4[1] = image[(0)*numCols + 1];
        arrayLocal_4[2] = image[(1)*numCols + 0], arrayLocal_4[3] = image[(1)*numCols + 1];
        OverlapPreFilter4(arrayLocal_4);
        image[(0)*numCols + 0] = arrayLocal_4[0], image[(0)*numCols + 1] = arrayLocal_4[1];
        image[(1)*numCols + 0] = arrayLocal_4[2], image[(1)*numCols + 1] = arrayLocal_4[3];

        // top right
        arrayLocal_4[0] = image[(0)*numCols + numCols-2], arrayLocal_4[1] = image[(0)*numCols + numCols-1];
        arrayLocal_4[2] = image[(1)*numCols + numCols-2], arrayLocal_4[3] = image[(1)*numCols + numCols-1];
        OverlapPreFilter4(arrayLocal_4);
        image[(0)*numCols + numCols-2] = arrayLocal_4[0], image[(0)*numCols + numCols-1] = arrayLocal_4[1];
        image[(1)*numCols + numCols-2] = arrayLocal_4[2], image[(1)*numCols + numCols-1] = arrayLocal_4[3];

        // bottom left
        arrayLocal_4[0] = image[(numRows-2)*numCols + 0], arrayLocal_4[1] = image[(numRows-2)*numCols + 1];
        arrayLocal_4[2] = image[(numRows-1)*numCols + 0], arrayLocal_4[3] = image[(numRows-1)*numCols + 1];
        OverlapPreFilter4(arrayLocal_4);
        image[(numRows-2)*numCols + 0] = arrayLocal_4[0], image[(numRows-2)*numCols + 1] = arrayLocal_4[1];
        image[(numRows-1)*numCols + 0] = arrayLocal_4[2], image[(numRows-1)*numCols + 1] = arrayLocal_4[3];

        // bottom right
        arrayLocal_4[0] = image[(numRows-2)*numCols + numCols-2], arrayLocal_4[1] = image[(numRows-2)*numCols + numCols-1];
        arrayLocal_4[2] = image[(numRows-1)*numCols + numCols-2], arrayLocal_4[3] = image[(numRows-1)*numCols + numCols-1];
        OverlapPreFilter4(arrayLocal_4);
        image[(numRows-2)*numCols + numCols-2] = arrayLocal_4[0], image[(numRows-2)*numCols + numCols-1] = arrayLocal_4[1];
        image[(numRows-1)*numCols + numCols-2] = arrayLocal_4[2], image[(numRows-1)*numCols + numCols-1] = arrayLocal_4[3];
    }
}


int main()
{
    struct timeval tim[50];
    int t = 0;

    FILE *time_log = fopen("timing_log.txt", "w");

    gettimeofday(&tim[t], NULL); t++;
    FILE *ip = fopen("BImage", "r");
    FILE *op = fopen("encoded.txt", "w");

    // read image in host
    int imageWidth = 16000, imageHeight=8000;
    //scanf("%d %d", &imageHeight, &imageWidth);
    //int image[128][112]; // = 
    int *image = (int*) malloc(imageHeight * imageWidth * sizeof(int) );
    int i, j;

    for(i = 0; i < imageHeight; i++){
        //image[i] = (int*) malloc(imageWidth * sizeof(int) );

        for(j = 0; j < imageWidth; j++)
            fscanf(ip, "%d", &image[i * imageWidth + j]);
    }
    gettimeofday(&tim[t], NULL); t++;

    // allocate & copy image memory in device
    int *imageDevice;
    size_t size = imageWidth * imageHeight * sizeof(int);
    hipMalloc((void**) &imageDevice, size );
    hipMemcpy(imageDevice, image, size, hipMemcpyHostToDevice);

    /* kernel invocation start*/
    dim3 DimGrid(imageHeight/16, imageWidth/16);
    dim3 DimBlock(4, 4);
    dim3 DimGrid2(imageHeight/4-1, imageWidth/4-1);
    dim3 DimGrid3(imageHeight/16-1, imageWidth/16-1);

    gettimeofday(&tim[t], NULL); t++;

    // second stage frequency transform
    EncSecondStageOverlapFilter<<< DimGrid2, 1>>>(imageDevice, imageHeight, imageWidth);
    gettimeofday(&tim[t], NULL); t++;

    // first stage pre-filtering
    EncFirstStagePreFiltering<<< DimGrid, DimBlock>>>(imageDevice, imageHeight, imageWidth);
    gettimeofday(&tim[t], NULL); t++;

    // first stage frequency transform
    EncFirstStageOverlapFilter<<< DimGrid3, 1>>>(imageDevice, imageHeight, imageWidth);
    gettimeofday(&tim[t], NULL); t++;

    // second stage pre-filtering
    EncSecondStagePreFiltering<<< DimGrid, 1>>>(imageDevice, imageHeight, imageWidth);
    gettimeofday(&tim[t], NULL); t++;

    /* kernel function invocation end*/
    hipDeviceSynchronize();
    gettimeofday(&tim[t], NULL); t++;

    // copy from device to host
    hipMemcpy(image, imageDevice, size, hipMemcpyDeviceToHost);

    //free device memory
    hipFree(imageDevice);
    gettimeofday(&tim[t], NULL); t++;

    //store processed image in file
    for( i = 0; i < imageHeight; i++)
    {
        for( j = 0; j < imageWidth; j++)
            fprintf(op, "%d ", image[i * imageWidth + j] );
        fprintf(op, "\n");
    }
    gettimeofday(&tim[t], NULL); t++;

    //time logs
    i = 0;

    fprintf(time_log, "Image Read from file: %lu seconds %lu microseconds\n", tim[i+1].tv_sec - tim[i].tv_sec, tim[i+1].tv_usec - tim[i].tv_usec);
    i++;
    fprintf(time_log, "Image Copy To Device: %lu seconds %lu microseconds\n", tim[i+1].tv_sec - tim[i].tv_sec, tim[i+1].tv_usec - tim[i].tv_usec);
    i++;
    fprintf(time_log, "Stage 1: %lu seconds %lu microseconds\n", tim[i+1].tv_sec - tim[i].tv_sec, tim[i+1].tv_usec - tim[i].tv_usec);
    i++;
    fprintf(time_log, "Stage 2: %lu seconds %lu microseconds\n", tim[i+1].tv_sec - tim[i].tv_sec, tim[i+1].tv_usec - tim[i].tv_usec);
    i++;
    fprintf(time_log, "Stage 3: %lu seconds %lu microseconds\n", tim[i+1].tv_sec - tim[i].tv_sec, tim[i+1].tv_usec - tim[i].tv_usec);
    i++;
    fprintf(time_log, "Stage 4: %lu seconds %lu microseconds\n", tim[i+1].tv_sec - tim[i].tv_sec, tim[i+1].tv_usec - tim[i].tv_usec);
    i++;
    fprintf(time_log, "Cuda Device Synchronise: %lu seconds %lu microseconds\n", tim[i+1].tv_sec - tim[i].tv_sec, tim[i+1].tv_usec - tim[i].tv_usec);
    i++;
    fprintf(time_log, "Image COpy back to Host: %lu seconds %lu microseconds\n", tim[i+1].tv_sec - tim[i].tv_sec, tim[i+1].tv_usec - tim[i].tv_usec);
    i++;
    fprintf(time_log, "Writing Encoded to File: %lu seconds %lu microseconds\n", tim[i+1].tv_sec - tim[i].tv_sec, tim[i+1].tv_usec - tim[i].tv_usec);

    fclose(ip);
    fclose(op);
    fclose(time_log);
    return 0;
}
