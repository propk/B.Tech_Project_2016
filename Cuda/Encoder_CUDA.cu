
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//__global__ int *arrayLocal_16, *arrayLocal_4;

__device__ void T2x2H(int *iCoeff, int valRound)
{
    int valT1, valT2;
    iCoeff[0] += iCoeff[3];
    iCoeff[1] -= iCoeff[2];
    valT1 = ((iCoeff[0] - iCoeff[1] + valRound) >> 1);
    valT2 = iCoeff[2];
    iCoeff[2] = valT1 - iCoeff[3];
    iCoeff[3] = valT1 - valT2;
    iCoeff[0] -= iCoeff[3];
    iCoeff[1] += iCoeff[2];
}

__device__ void TOdd(int *iCoeff, int dummy)
{
    iCoeff[1] -= iCoeff[2];
    iCoeff[0] += iCoeff[3];
    iCoeff[2] += ((iCoeff[1] + 1) >> 1);
    iCoeff[3] = ((iCoeff[0] + 1) >> 1) - iCoeff[3];
    iCoeff[1] -= ((3* iCoeff[0] + 4) >> 3);
    iCoeff[0] += ((3* iCoeff[1] + 4) >> 3);
    iCoeff[3] -= ((3* iCoeff[2] + 4) >> 3);
    iCoeff[2] += ((3* iCoeff[3] + 4) >> 3);
    iCoeff[3] += (iCoeff[1] >> 1);
    iCoeff[2] -= ((iCoeff[0] + 1) >> 1);
    iCoeff[1] -= iCoeff[3];
    iCoeff[0] += iCoeff[2];
}

__device__ void TOddOdd(int *iCoeff, int dummy)
{
    int valT1, valT2;
    iCoeff[1] = -iCoeff[1];
    iCoeff[2] = -iCoeff[2];
    iCoeff[3] += iCoeff[0];
    iCoeff[2] -= iCoeff[1];
    iCoeff[0] -= (valT1 = iCoeff[3] >> 1);
    iCoeff[1] += (valT2 = iCoeff[2] >> 1);
    iCoeff[0] += ((iCoeff[1] * 3 + 4) >> 3);
    iCoeff[1] -= ((iCoeff[0] * 3 + 3) >> 2);
    iCoeff[0] += ((iCoeff[1] * 3 + 3) >> 3);
    iCoeff[1] -= valT2;
    iCoeff[0] += valT1;
    iCoeff[2] += iCoeff[1];
    iCoeff[3] -= iCoeff[0];
}

__device__ void FwdPermute(int *arrayInput)
{
    int i;
    int arrayTemp[16];
    int FwdPermArr[16] = {
        0, 8, 4, 6, 2, 10, 14, 12,
        1, 11, 15, 13, 9, 3, 7, 5
    };
    for (i = 0; i <= 15; i++)
        arrayTemp[FwdPermArr[i]] = arrayInput[i];
    for (i = 0; i <= 15; i++)
        arrayInput[i] = arrayTemp[i];
}

__device__ void (*pointerFunct_FCT[8]) (int *arg1, int arg2) = {
    T2x2H, T2x2H, T2x2H, T2x2H,
    T2x2H, TOdd, TOdd, TOddOdd
};
__device__ void FCT4x4(int *iCoeff)
{
    int arrayLocal[4];
    int arrayTemp[8][4] = {
        { 0, 3, 12, 15},
        { 5, 6, 9, 10},
        { 1, 2, 13, 14},
        { 4, 7, 8, 11},
        { 0, 1, 4, 5},
        { 2, 3, 6, 7},
        { 8, 12, 9, 13},
        { 10, 11, 14, 15}
    };


    int arg2Array[8] = { 0, 0, 0, 0, 1, 0, 0, 0};
    int i, j;

    for(i = 0; i < 8; i++)
    {
        for(j = 0; j < 4; j++)
            arrayLocal[j] = iCoeff[arrayTemp[i][j]];

        (*pointerFunct_FCT[i]) (arrayLocal, arg2Array[i]);

        for(j = 0; j < 4; j++)
            iCoeff[arrayTemp[i][j]] = arrayLocal[j];
    }

    FwdPermute(iCoeff);
}

__device__ void T2x2hEnc(int *iCoeff, int dummy)
{
    int valT1, valT2;
    iCoeff[0] += iCoeff[3];
    iCoeff[1] -= iCoeff[2];
    valT1 = iCoeff[3];
    valT2 = iCoeff[2];
    iCoeff[2] = ((iCoeff[0] - iCoeff[1]) >> 1) - valT1;
    iCoeff[3] = valT2 + (iCoeff[1] >> 1);
    iCoeff[1] += iCoeff[2];
    iCoeff[0] -= (iCoeff[3] * 3 + 4) >> 3;
}

__device__ void FwdRotate(int *iCoeff, int dummy)
{
    iCoeff[1] -= (iCoeff[0] + 1) >> 1;
    iCoeff[0] += (iCoeff[1] + 1) >> 1;
}

__device__ void FwdScale(int *iCoeff, int dummy)
{
    iCoeff[1] -= (iCoeff[0] * 3 + 0) >> 4;
    iCoeff[1] -= (iCoeff[0] >> 7);
    iCoeff[1] += (iCoeff[0] >> 10);
    iCoeff[0] -= (iCoeff[1] * 3 + 0) >> 3;
    iCoeff[1] = (iCoeff[0] >> 1) - iCoeff[1];
    iCoeff[0] -= iCoeff[1];
}

__device__ void FwdTOddOdd(int *iCoeff, int dummy)
{
    int valT1, valT2;
    iCoeff[3] += iCoeff[0];
    iCoeff[2] -= iCoeff[1];
    valT1 = iCoeff[3] >> 1;
    valT2 = iCoeff[2] >> 1;
    iCoeff[0] -= valT1;
    iCoeff[1] += valT2;
    iCoeff[0] += (iCoeff[1] * 3 + 4) >>3;
    iCoeff[1] -= (iCoeff[0] * 3 + 2) >> 2;
    iCoeff[0] += (iCoeff[1] * 3 + 6) >> 3;
    iCoeff[1] -= valT2;
    iCoeff[0] += valT1;
    iCoeff[2] += iCoeff[1];
    iCoeff[3] -= iCoeff[0];
}

__device__ void (*pointerFunct[17]) (int *arg1, int arg2) = {
    T2x2hEnc, T2x2hEnc, T2x2hEnc, T2x2hEnc,
    FwdScale, FwdScale, FwdScale, FwdScale,
    FwdRotate, FwdRotate, FwdRotate, FwdRotate,
    FwdTOddOdd, T2x2H, T2x2H, T2x2H, T2x2H
};

__device__ void OverlapPreFilter4x4(int *iCoeff)
{
    int arrayLocal[4];
    int arrayTemp[17][4] = {
        { 0, 3, 12, 15},
        { 1, 2, 13, 14},
        { 4, 7, 8, 11},
        { 5, 6, 9, 10},

        { 0, 15, -1, -1},
        { 1, 14, -1, -1},
        { 4, 11, -1, -1},
        { 5, 10, -1, -1},

        { 13, 12, -1, -1},
        { 9, 8, -1, -1},
        { 7, 3, -1, -1},
        { 6, 2, -1, -1},

        { 10, 11, 14, 15},
        { 0, 12, 3, 15},
        { 1, 2, 13, 14},
        { 4, 7, 8, 11},
        { 5, 6, 9, 10}
    };


    int i, j;

    for(i = 0; i < 17; i++)
    {
        for(j = 0; j < 4; j++)
            if(arrayTemp[i][j] >= 0)
                arrayLocal[j] = iCoeff[arrayTemp[i][j]];

        (*pointerFunct[i]) (arrayLocal, 0);

        for(j = 0; j < 4; j++)
            if(arrayTemp[i][j] >= 0)
                iCoeff[arrayTemp[i][j]] = arrayLocal[j];
    }
}

__device__ void OverlapPreFilter4(int *iCoeff)
{
    int arrayLocal[2];
    iCoeff[0] += iCoeff[3];
    iCoeff[1] += iCoeff[2];
    iCoeff[3] -= ((iCoeff[0] + 1) >> 1);
    iCoeff[2] -= ((iCoeff[1] + 1) >> 1);
    arrayLocal[0] = iCoeff[2], arrayLocal[1] = iCoeff[3];
    FwdRotate(arrayLocal, 0);
    iCoeff[2] = arrayLocal[0], iCoeff[3] = arrayLocal[1];
    iCoeff[3] = -iCoeff[3];
    iCoeff[2] = -iCoeff[2];
    iCoeff[0] -= iCoeff[3];
    iCoeff[1] -= iCoeff[2];
    iCoeff[3] += ( iCoeff[0] >> 1);
    iCoeff[2] += ( iCoeff[1] >> 1);
    iCoeff[0] -= ((iCoeff[3] * 3 + 4) >> 3);
    iCoeff[1] -= ((iCoeff[2] * 3 + 4) >> 3);
    arrayLocal[0] = iCoeff[0], arrayLocal[1] = iCoeff[3];
    FwdScale(arrayLocal, 0);
    iCoeff[0] = arrayLocal[0], iCoeff[3] = arrayLocal[1];

    arrayLocal[0] = iCoeff[1], arrayLocal[1] = iCoeff[2];
    FwdScale(arrayLocal, 0);
    iCoeff[1] = arrayLocal[0], iCoeff[2] = arrayLocal[1];
    iCoeff[3] += ((iCoeff[0] + 1) >> 1);
    iCoeff[2] += ((iCoeff[1] + 1) >> 1);
    iCoeff[0] -= iCoeff[3];
    iCoeff[1] -= iCoeff[2];
}

__device__ void OverlapPreFilter2x2(int *iCoeff)
{
    iCoeff[0] += iCoeff[3];
    iCoeff[1] += iCoeff[2];
    iCoeff[3] -= ((iCoeff[0] + 1) >> 1);
    iCoeff[2] -= ((iCoeff[1] + 1) >> 1);
    iCoeff[1] -= ((iCoeff[0] + 2) >> 2);
    iCoeff[0] -= (iCoeff[1] >> 5);
    iCoeff[0] -= (iCoeff[1] >> 9);
    iCoeff[0] -= (iCoeff[1] >> 13);
    iCoeff[0] -= ((iCoeff[1] + 1) >> 1);
    iCoeff[1] -= ((iCoeff[0] + 2) >> 2);
    iCoeff[3] += ((iCoeff[0] + 1) >> 1);
    iCoeff[2] += ((iCoeff[1] + 1) >> 1);
    iCoeff[0] -= iCoeff[3];
    iCoeff[1] -= iCoeff[2];
}

__device__ void OverlapPreFilter2(int *iCoeff)
{
    iCoeff[1] -= ((iCoeff[0] + 2) >> 2);
    iCoeff[0] -= (iCoeff[1] >> 13);
    iCoeff[0] -= (iCoeff[1] >> 9);
    iCoeff[0] -= (iCoeff[1] >> 5);
    iCoeff[0] -= ((iCoeff[1] + 1) >> 1);
    iCoeff[1] -= ((iCoeff[0] + 2) >> 2);
}

__global__ void EncFirstStagePreFiltering(int *image, int numRows, int numCols)
{
    int i, j;
    int arrayLocal[16];
    int block_i = threadIdx.x, block_j = threadIdx.y;
    int macro_i = blockIdx.x, macro_j = blockIdx.y;

    for (i = 0; i < 4; ++i)
    {
        for (j = 0; j < 4; ++j)
        {
            arrayLocal[i*4 + j] = image[ (macro_i*16 + block_i*4 + i) * numCols + macro_j*16 + block_j*4 + j];
        }
    }

    FCT4x4(arrayLocal);

    for (i = 0; i < 4; ++i)
    {
        for (j = 0; j < 4; ++j)
        {
            image[(macro_i*16 + block_i*4 + i) * numCols + macro_j*16 + block_j*4 + j] = arrayLocal[i*4 + j];
        }
    }
}

__global__ void EncSecondStagePreFiltering(int* image, int numRows, int numCols)
{
    int i, j;
    int arrayLocal[16];
    int macro_i = blockIdx.x, macro_j = blockIdx.y;

    for (i = 0; i < 4; ++i)
    {
        for (j = 0; j < 4; ++j)
        {
            arrayLocal[i*4 + j] = image[ (macro_i*16 + i*4) * numCols + macro_j*16 + j*4];
        }
    }
    FCT4x4(arrayLocal);
    for (i = 0; i < 4; ++i)
    {
        for (j = 0; j < 4; ++j)
        {
            image[(macro_i*16 + i*4) * numCols + macro_j*16 + j*4] = arrayLocal[i*4 + j];
        }
    }
}

__global__ void EncFirstStageOverlapFilter(int* image, int numRows, int numCols)
{
    int arrayLocal_16[16], arrayLocal_4[4];
    int block_i = blockIdx.x, block_j = blockIdx.y, i, j;

    //numRows /= 4;
    //numCols /= 4;
/*
    // 4x4 blocks
    for( i = 0; i < 4; i++)
        for( j = 0; j < 4; j++)
            arrayLocal_16[i*4+j] = image[((block_i + i)*4)*numCols + (block_j + j)*4];

    OverlapPreFilter4x4(arrayLocal_16);
    for( i = 0; i < 4; i++)
        for( j = 0; j < 4; j++)
            image[((block_i + i)*4)*numCols + (block_j + j)*4] = arrayLocal_16[i*4+j];
    //4x4 block end
*/
    if(block_j == 0)
    {
        //left edge
        for(i = 0; i < 2; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[((block_i*4+j+2)*4)*numCols + i*4];
            OverlapPreFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[((block_i*4+j+2)*4)*numCols + i*4] = arrayLocal_4[j];
        }
        // right edge
        for(i = numCols-2; i < numCols; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[((block_i*4+j+2)*4)*numCols + i*4];
            OverlapPreFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[((block_i*4+j+2)*4)*numCols + i*4] = arrayLocal_4[j];
        }
    }


    if(block_i == 0)
    {
        //top edge
        for(i = 0; i < 2; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[(i*4)*numCols + (block_j*4+j+2)*4];
            OverlapPreFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[(i*4)*numCols + (block_j*4+j+2)*4] = arrayLocal_4[j];
        }
        //bottom edge
        for(i = numRows-2; i < numRows; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[(i*4)*numCols + (block_j*4+2+j)*4];
            OverlapPreFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[(i*4)*numCols + (block_j*4+j+2)*4] = arrayLocal_4[j];
        }
    }

    if(block_j == 0 && block_i == 0)
    {
        // top left
        arrayLocal_4[0] = image[(0)*numCols + 0], arrayLocal_4[1] = image[(0)*numCols + 4];
        arrayLocal_4[2] = image[(4)*numCols + 0], arrayLocal_4[3] = image[(4)*numCols + 4];
        OverlapPreFilter4(arrayLocal_4);
        image[(0)*numCols + 0] = arrayLocal_4[0], image[(0)*numCols + 4] = arrayLocal_4[1];
        image[(4)*numCols + 0] = arrayLocal_4[2], image[(4)*numCols + 4] = arrayLocal_4[3];

        // top right
        arrayLocal_4[0] = image[(0)*numCols + (numCols-2)*4], arrayLocal_4[1] = image[(0)*numCols + (numCols-1)*4];
        arrayLocal_4[2] = image[(4)*numCols + (numCols-2)*4], arrayLocal_4[3] = image[(4)*numCols + (numCols-1)*4];
        OverlapPreFilter4(arrayLocal_4);
        image[(0)*numCols + (numCols-2)*4] = arrayLocal_4[0], image[(0)*numCols + (numCols-1)*4] = arrayLocal_4[1];
        image[(4)*numCols + (numCols-2)*4] = arrayLocal_4[2], image[(4)*numCols + (numCols-1)*4] = arrayLocal_4[3];

        // bottom left
        arrayLocal_4[0] = image[((numRows-2)*4)*numCols + 0], arrayLocal_4[1] = image[((numRows-2)*4)*numCols + 4];
        arrayLocal_4[2] = image[((numRows-1)*4)*numCols + 0], arrayLocal_4[3] = image[((numRows-1)*4)*numCols + 4];
        OverlapPreFilter4(arrayLocal_4);
        image[((numRows-2)*4)*numCols + 0] = arrayLocal_4[0], image[((numRows-2)*4)*numCols + 4] = arrayLocal_4[1];
        image[((numRows-1)*4)*numCols + 0] = arrayLocal_4[2], image[((numRows-1)*4)*numCols + 4] = arrayLocal_4[3];

        // bottom right
        arrayLocal_4[0] = image[((numRows-2)*4)*numCols + (numCols-2)*4], arrayLocal_4[1] = image[((numRows-2)*4)*numCols + (numCols-1)*4];
        arrayLocal_4[2] = image[((numRows-1)*4)*numCols + (numCols-2)*4], arrayLocal_4[3] = image[((numRows-1)*4)*numCols + (numCols-1)*4];
        OverlapPreFilter4(arrayLocal_4);
        image[((numRows-2)*4)*numCols + (numCols-2)*4] = arrayLocal_4[0], image[((numRows-2)*4)*numCols + (numCols-1)*4] = arrayLocal_4[1];
        image[((numRows-1)*4)*numCols + (numCols-2)*4] = arrayLocal_4[2], image[((numRows-1)*4)*numCols + (numCols-1)*4] = arrayLocal_4[3];
    }
}

__global__ void EncSecondStageOverlapFilter(int* image, int numRows, int numCols)
{
    int arrayLocal_16[16], arrayLocal_4[4];
    int block_i = blockIdx.x, block_j = blockIdx.y, i, j;

    // 4x4 blocks
    for( i = 0; i < 4; i++)
    {
        for( j = 0; j < 4; j++)
            arrayLocal_16[i*4+j] = image[ (block_i*4 + 2 + i)*numCols + block_j*4 + 2 + j];
    }
    OverlapPreFilter4x4(arrayLocal_16);
    for( i = 0; i < 4; i++)
    {
        for( j = 0; j < 4; j++)
            image[(block_i*4 + 2 + i)*numCols + block_j*4 + 2 + j] = arrayLocal_16[i*4+j];
    }
    //4x4 block end

    if(block_j == 0)
    {
        //left edge
        for(i = 0; i < 2; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[(block_i*4 + 2+j)*numCols + i];
            OverlapPreFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[(block_i*4 + 2+j)*numCols + i] = arrayLocal_4[j];
        }
        // right edge
        for(i = numCols-2; i < numCols; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[(block_i*4 + 2 + j)*numCols + i];
            OverlapPreFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[(block_i*4 + 2+j)*numCols + i] = arrayLocal_4[j];
        }
    }


    if(block_i == 0)
    {
        //top edge
        for(i = 0; i < 2; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[(i)*numCols + block_j*4 + 2+j];
            OverlapPreFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[(i)*numCols + block_j*4 + 2+j] = arrayLocal_4[j];
        }
        //bottom edge
        for(i = numRows-2; i < numRows; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[(i)*numCols + block_j*4 + 2 + j];
            OverlapPreFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[(i)*numCols + block_j*4 + 2 + j] = arrayLocal_4[j];
        }
    }

    if(block_i == 0 && block_j == 0)
    {
        // top left
        arrayLocal_4[0] = image[(0)*numCols + 0], arrayLocal_4[1] = image[(0)*numCols + 1];
        arrayLocal_4[2] = image[(1)*numCols + 0], arrayLocal_4[3] = image[(1)*numCols + 1];
        OverlapPreFilter4(arrayLocal_4);
        image[(0)*numCols + 0] = arrayLocal_4[0], image[(0)*numCols + 1] = arrayLocal_4[1];
        image[(1)*numCols + 0] = arrayLocal_4[2], image[(1)*numCols + 1] = arrayLocal_4[3];

        // top right
        arrayLocal_4[0] = image[(0)*numCols + numCols-2], arrayLocal_4[1] = image[(0)*numCols + numCols-1];
        arrayLocal_4[2] = image[(1)*numCols + numCols-2], arrayLocal_4[3] = image[(1)*numCols + numCols-1];
        OverlapPreFilter4(arrayLocal_4);
        image[(0)*numCols + numCols-2] = arrayLocal_4[0], image[(0)*numCols + numCols-1] = arrayLocal_4[1];
        image[(1)*numCols + numCols-2] = arrayLocal_4[2], image[(1)*numCols + numCols-1] = arrayLocal_4[3];

        // bottom left
        arrayLocal_4[0] = image[(numRows-2)*numCols + 0], arrayLocal_4[1] = image[(numRows-2)*numCols + 1];
        arrayLocal_4[2] = image[(numRows-1)*numCols + 0], arrayLocal_4[3] = image[(numRows-1)*numCols + 1];
        OverlapPreFilter4(arrayLocal_4);
        image[(numRows-2)*numCols + 0] = arrayLocal_4[0], image[(numRows-2)*numCols + 1] = arrayLocal_4[1];
        image[(numRows-1)*numCols + 0] = arrayLocal_4[2], image[(numRows-1)*numCols + 1] = arrayLocal_4[3];

        // bottom right
        arrayLocal_4[0] = image[(numRows-2)*numCols + numCols-2], arrayLocal_4[1] = image[(numRows-2)*numCols + numCols-1];
        arrayLocal_4[2] = image[(numRows-1)*numCols + numCols-2], arrayLocal_4[3] = image[(numRows-1)*numCols + numCols-1];
        OverlapPreFilter4(arrayLocal_4);
        image[(numRows-2)*numCols + numCols-2] = arrayLocal_4[0], image[(numRows-2)*numCols + numCols-1] = arrayLocal_4[1];
        image[(numRows-1)*numCols + numCols-2] = arrayLocal_4[2], image[(numRows-1)*numCols + numCols-1] = arrayLocal_4[3];
    }
}


int main()
{
    FILE *ip = fopen("image.txt", "r");
    FILE *op = fopen("encoded.txt", "w");
    //printf("1\n");
    // read image in host
    int imageWidth = 112, imageHeight=128;
    //scanf("%d %d", &imageHeight, &imageWidth);
    int image[128][112]; // = (int**) malloc(imageHeight * sizeof(int*) );
    int i, j;

    for(i = 0; i < imageHeight; i++){
        //image[i] = (int*) malloc(imageWidth * sizeof(int) );

        for(j = 0; j < imageWidth; j++)
            fscanf(ip, "%d", &image[i][j]);
    }

    // allocate & copy image memory in device
    int *imageDevice;
    size_t size = imageWidth * imageHeight * sizeof(int);
    hipMalloc((void**) &imageDevice, size );
    hipMemcpy(imageDevice, image, size, hipMemcpyHostToDevice);

    /* kernel invocation start*/
    dim3 DimGrid(imageHeight/16, imageWidth/16);
    dim3 DimBlock(4, 4);
    dim3 DimGrid2(imageHeight/4-1, imageWidth/4-1);
    dim3 DimGrid3(imageHeight/16-1, imageWidth/16-1);
    // second stage frequency transform
    //EncSecondStageOverlapFilter<<< DimGrid2, 1>>>(imageDevice, imageHeight, imageWidth);
    // first stage pre-filtering
    //EncFirstStagePreFiltering<<< DimGrid, DimBlock>>>(imageDevice, imageHeight, imageWidth);
    // first stage frequency transform
    EncFirstStageOverlapFilter<<< DimGrid3, 1>>>(imageDevice, imageHeight, imageWidth);
    // second stage pre-filtering
    //EncSecondStagePreFiltering<<< DimGrid, 1>>>(imageDevice, imageHeight, imageWidth);

    /* kernel function invocation end*/
    hipDeviceSynchronize();
    // copy from device to host
    hipMemcpy(image, imageDevice, size, hipMemcpyDeviceToHost);


    //free device memory
    hipFree(imageDevice);

    //store processed image in file
    for( i = 0; i < imageHeight; i++)
    {
        for( j = 0; j < imageWidth; j++)
            fprintf(op, "%d ", image[i][j] );
        fprintf(op, "\n");
    }
    printf("Completed");
    fclose(ip);
    fclose(op);

    return 0;
}
