// MP 1

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
	int i = threadIdx.x +blockDim.x*blockIdx.x;
	if (i < len) out[i] = in1[i] + in2[i];
}

int main() {
  int inputLength = 4;
  float hostInput1[] = {1,2,3,4};
  float hostInput2[] = {5,6,7,8};
  float hostOutput[] = {0,0,0,0};
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;

  //@@ Allocate GPU memory here
	
	hipMalloc((void**) & deviceInput1, inputLength * sizeof(float));
	hipMalloc((void**) & deviceInput2, inputLength * sizeof(float));
	hipMalloc((void**) & deviceOutput, inputLength * sizeof(float));
	
  //@@ Copy memory to the GPU here
	
	hipMemcpy(deviceInput1, hostInput1, inputLength * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(deviceInput2, hostInput2, inputLength * sizeof(float), hipMemcpyHostToDevice);

  //@@ Initialize the grid and block dimensions here
	
	dim3 DimGrid((inputLength-1)/256+1, 1, 1);
	dim3 DimBlock(256, 1, 1);

  //@@ Launch the GPU Kernel here
	
	vecAdd<<< DimGrid, DimBlock >>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

  hipDeviceSynchronize();
  //@@ Copy the GPU memory back to the CPU here
	
	hipMemcpy(hostOutput, deviceOutput, inputLength * sizeof(float), hipMemcpyDeviceToHost);

  //@@ Free the GPU memory here
	
	hipFree(deviceInput1);
	hipFree(deviceInput2);
	hipFree(deviceOutput);
	
	printf("%lf, %lf, %lf, %lf", hostOutput[0], hostOutput[1], hostOutput[2], hostOutput[3]);

  return 0;
}
