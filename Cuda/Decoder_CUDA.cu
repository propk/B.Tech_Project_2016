#include "hip/hip_runtime.h"
#include<stdio.h>
#include <stdlib.h>
__device__ void T2x2H(int *iCoeff, int valRound)
{
    int valT1, valT2;
    iCoeff[0] += iCoeff[3];
    iCoeff[1] -= iCoeff[2];
    valT1 = ((iCoeff[0] - iCoeff[1] + valRound) >> 1);
    valT2 = iCoeff[2];
    iCoeff[2] = valT1 - iCoeff[3];
    iCoeff[3] = valT1 - valT2;
    iCoeff[0] -= iCoeff[3];
    iCoeff[1] += iCoeff[2];
}

__device__ void InvTOdd(int *iCoeff, int dummy)
{
    iCoeff[1] += iCoeff[3];
    iCoeff[0] -= iCoeff[2];
    iCoeff[3] -= (iCoeff[1] >> 1);
    iCoeff[2] += ((iCoeff[0] + 1) >> 1);
    iCoeff[0] -= ((3* iCoeff[1] + 4) >> 3);
    iCoeff[1] += ((3* iCoeff[0] + 4) >> 3);
    iCoeff[2] -= ((3* iCoeff[3] + 4) >> 3);
    iCoeff[3] += (3* iCoeff[2] + 4) >> 3;
    iCoeff[2] -= ((iCoeff[1] + 1) >> 1);
    iCoeff[3] = ((iCoeff[0] + 1) >> 1) - iCoeff[3];
    iCoeff[1] += iCoeff[2];
    iCoeff[0] -= iCoeff[3];
}

__device__ void InvTOddOdd(int *iCoeff, int dummy)
{
    int valT1, valT2;
    iCoeff[3] += iCoeff[0];
    iCoeff[2] -= iCoeff[1];
    valT1 = iCoeff[3] >> 1;
    valT2 = iCoeff[2] >> 1;
    iCoeff[0] -= valT1;
    iCoeff[1] += valT2;
    iCoeff[0] -= ((iCoeff[1] * 3 + 3) >> 3);
    iCoeff[1] += ((iCoeff[0] * 3 + 3) >> 2);
    iCoeff[0] -= ((iCoeff[1] * 3 + 4) >> 3);
    iCoeff[1] -= valT2;
    iCoeff[0] += valT1;
    iCoeff[2] += iCoeff[1];
    iCoeff[3] -= iCoeff[0];
    iCoeff[1] = -iCoeff[1];
    iCoeff[2] = -iCoeff[2];
}

__device__ void InvPermute(int *arrayInput)
{
    int i;
    int arrayTemp[16];
    int InvPermArr[16] = {
        0, 8, 4, 13, 2, 15, 3, 14,
        1, 12, 5, 9, 7, 11, 6, 10
    };
    for (i = 0; i <= 15; i++)
        arrayTemp[InvPermArr[i]] = arrayInput[i];
    for (i = 0; i <= 15; i++)
        arrayInput[i] = arrayTemp[i];
}

__device__ void InvPermute2pt(int *arrayInput)
{
    int arrayTemp[2], i;
    arrayTemp[0] = arrayInput[1];
    arrayTemp[1] = arrayInput[0];
    for (i = 0; i <= 1; i++)
        arrayInput[i] = arrayTemp[i];
}

__device__ void T2pt(int *iCoeff)
{
    iCoeff[0] -= (iCoeff[1]+1) >> 1;
    iCoeff[1] += iCoeff[0];
}

__device__ void (*pointerFunct[8]) (int *arg1, int arg2) = {
    T2x2H, InvTOdd, InvTOdd, InvTOddOdd,
    T2x2H, T2x2H, T2x2H, T2x2H
};

__device__ void ICT4x4(int *iCoeff)
{
    InvPermute(iCoeff);
    int arrayLocal[4];
    int arrayTemp[8][4] = {
        { 0, 1, 4, 5},
        { 2, 3, 6, 7},
        { 8, 12, 9, 13},
        { 10, 11, 14, 15},
        { 0, 3, 12, 15},
        { 5, 6, 9, 10},
        { 1, 2, 13, 14},
        { 4, 7, 8, 11}
    };


    int arg2Array[8] = { 1, 0, 0, 0, 0, 0, 0, 0};
    int i, j;

    for(i = 0; i < 8; i++)
    {
        for(j = 0; j < 4; j++)
            arrayLocal[j] = iCoeff[arrayTemp[i][j]];

        (*pointerFunct[i]) (arrayLocal, arg2Array[i]);

        for(j = 0; j < 4; j++)
            iCoeff[arrayTemp[i][j]] = arrayLocal[j];
    }
}

__device__ void InvTOddOddPOST(int* iCoeff, int dummy)
{
	int valT1, valT2;
	iCoeff[3] += iCoeff[0];
	iCoeff[2] -= iCoeff[1];
	valT1 = iCoeff[3] >> 1;
	valT2 = iCoeff[2] >> 1;
	iCoeff[0] -= valT1;
	iCoeff[1] += valT2;
	iCoeff[0] -= (iCoeff[1] * 3 + 6) >> 3;
	iCoeff[1] += (iCoeff[0] * 3 + 2) >> 2;
	iCoeff[0] -= (iCoeff[1] * 3 + 4) >> 3;
	iCoeff[1] -= valT2;
	iCoeff[0] += valT1;
	iCoeff[2] += iCoeff[1];
	iCoeff[3] -= iCoeff[0];
}

__device__ void T2x2HPOST(int* iCoeff, int dummy)
{
	int valT1;
	iCoeff[1] -= iCoeff[2];
	iCoeff[0] += (iCoeff[3] * 3 + 4) >> 3;
	iCoeff[3] -= (iCoeff[1] >> 1);
	iCoeff[2] = ((iCoeff[0] - iCoeff[1]) >> 1) - iCoeff[2];
	valT1 = iCoeff[2];
	iCoeff[2] = iCoeff[3];
	iCoeff[3] = valT1;
	iCoeff[0] -= iCoeff[3];
	iCoeff[1] += iCoeff[2];
}

__device__ void InvScale (int* iCoeff, int dummy)
{
	iCoeff[0] += iCoeff[1];
	iCoeff[1] = (iCoeff[0] >> 1) - iCoeff[1];
	iCoeff[0] += (iCoeff[1] * 3 + 0) >> 3;
	iCoeff[1] += (iCoeff[0] * 3 + 0) >> 4;
	iCoeff[1] += (iCoeff[0] >> 7);
	iCoeff[1] -= (iCoeff[0] >> 10);
}

__device__ void InvRotate(int* iCoeff, int dummy)
{
	iCoeff[0] -= ((iCoeff[1] + 1) >> 1);
	iCoeff[1] += ((iCoeff[0] + 1) >> 1);
}

__device__ void OverlapPostFilter2(int* iCoeff)
{
	iCoeff[1] += ((iCoeff[0] + 2) >> 2);
	iCoeff[0] += ((iCoeff[1] + 1) >> 1);
	iCoeff[0] += (iCoeff[1] >> 5);
	iCoeff[0] += (iCoeff[1] >> 9);
	iCoeff[0] += (iCoeff[1] >> 13);
	iCoeff[1] += ((iCoeff[0] + 2) >> 2);
}

__device__ void OverlapPostFilter2x2(int* iCoeff)
{
	iCoeff[0] += iCoeff[3];
	iCoeff[1] += iCoeff[2];
	iCoeff[3] -= ((iCoeff[0] + 1) >> 1);
	iCoeff[2] -= ((iCoeff[1] + 1) >> 1);
	iCoeff[1] += ((iCoeff[0] + 2) >> 2);
	iCoeff[0] += ((iCoeff[1] + 1) >> 1);
	iCoeff[0] += (iCoeff[1] >> 5);
	iCoeff[0] += (iCoeff[1] >> 9);
	iCoeff[0] += (iCoeff[1] >> 13);
	iCoeff[1] += ((iCoeff[0] + 2) >> 2);
	iCoeff[3] += ((iCoeff[0] + 1) >> 1);
	iCoeff[2] += ((iCoeff[1] + 1) >> 1);
	iCoeff[0] -= iCoeff[3];
	iCoeff[1] -= iCoeff[2];
}

__device__ void OverlapPostFilter4(int *iCoeff)
{
    int arrayLocal[2];
	iCoeff[0] += iCoeff[3];
	iCoeff[1] += iCoeff[2];
	iCoeff[3] -= ((iCoeff[0] + 1) >> 1);
	iCoeff[2] -= ((iCoeff[1] + 1) >> 1);
	arrayLocal[0] = iCoeff[0], arrayLocal[1] = iCoeff[3];
	InvScale(arrayLocal, 0);
	iCoeff[0] = arrayLocal[0], iCoeff[3] = arrayLocal[1];

	arrayLocal[0] = iCoeff[1], arrayLocal[1] = iCoeff[2];
	InvScale(arrayLocal, 0);
	iCoeff[1] = arrayLocal[0], iCoeff[2] = arrayLocal[1];
	iCoeff[0] += ((iCoeff[3] * 3+ 4) >> 3);
	iCoeff[1] += ((iCoeff[2] * 3 + 4) >> 3);
	iCoeff[3] -= ( iCoeff[0] >> 1);
	iCoeff[2] -= ( iCoeff[1] >> 1);
	iCoeff[0] += iCoeff[3];
	iCoeff[1] += iCoeff[2];
	iCoeff[3] = -iCoeff[3];
	iCoeff[2] = -iCoeff[2];
	arrayLocal[0] = iCoeff[2], arrayLocal[1] = iCoeff[3];
	InvRotate(arrayLocal, 0);
	iCoeff[2] = arrayLocal[0], iCoeff[3] = arrayLocal[1];
	iCoeff[3] += ((iCoeff[0] + 1) >> 1);
	iCoeff[2] += ((iCoeff[1] + 1) >> 1);
	iCoeff[0] -= iCoeff[3];
	iCoeff[1] -= iCoeff[2];
}

__device__ void (*pointerFunct[17]) (int *arg1, int arg2) = {
    T2x2H, T2x2H, T2x2H, T2x2H,
    InvRotate, InvRotate, InvRotate, InvRotate,
    InvTOddOddPOST, InvScale, InvScale, InvScale, InvScale,
    T2x2HPOST, T2x2HPOST, T2x2HPOST, T2x2HPOST
};

__device__ void OverlapPostFilter4x4(int *iCoeff)
{
	int arrayLocal[4];
    int arrayTemp[17][4] = {
        { 0, 3, 12, 15},
        { 1, 2, 13, 14},
        { 4, 7, 8, 11},
        { 5, 6, 9, 10},

        { 13, 12, -1, -1},
        { 9, 8, -1, -1},
        { 7, 3, -1, -1},
        { 6, 2, -1, -1},

        { 10, 11, 14, 15},

        { 0, 15, -1, -1},
        { 1, 14, -1, -1},
        { 4, 11, -1, -1},
        { 5, 10, -1, -1},


        { 0, 3, 12, 15},
        { 1, 2, 13, 14},
        { 4, 7, 8, 11},
        { 5, 6, 9, 10}
    };


    int i, j;

    for(i = 0; i < 17; i++)
    {
        for(j = 0; j < 4; j++)
            if(arrayTemp[i][j] >= 0)
                arrayLocal[j] = iCoeff[arrayTemp[i][j]];

        (*pointerFunct[i]) (arrayLocal, 0);

        for(j = 0; j < 4; j++)
            if(arrayTemp[i][j] >= 0)
                iCoeff[arrayTemp[i][j]] = arrayLocal[j];
    }
}

__global__ void DecFirstStagePostFiltering(int *image, int numRows, int numCols)
{
    int i, j;
    int arrayLocal[16];
    int block_i = threadIdx.x, block_j = threadIdx.y;
    int macro_i = blockIdx.x, macro_j = blockIdx.y;

    for (i = 0; i < 4; ++i)
    {
        for (j = 0; j < 4; ++j)
        {
            arrayLocal[i*4 + j] = image[ (macro_i*16 + block_i*4 + i) * numCols + macro_j*16 + block_j*4 + j];
        }
    }

    ICT4x4(arrayLocal);

    for (i = 0; i < 4; ++i)
    {
        for (j = 0; j < 4; ++j)
        {
            image[(macro_i*16 + block_i*4 + i) * numCols + macro_j*16 + block_j*4 + j] = arrayLocal[i*4 + j];
        }
    }
}

__global__ void DecSecondStagePostFiltering(int * image, int numRows, int numCols)
{
    int i, j;
    int arrayLocal[16];
    int macro_i = blockIdx.x, macro_j = blockIdx.y;

    for (i = 0; i < 4; ++i)
    {
        for (j = 0; j < 4; ++j)
        {
            arrayLocal[i*4 + j] = image[ (macro_i*16 + i*4) * numCols + macro_j*16 + j*4];
        }
    }
    ICT4x4(arrayLocal);
    for (i = 0; i < 4; ++i)
    {
        for (j = 0; j < 4; ++j)
        {
            image[(macro_i*16 + i*4) * numCols + macro_j*16 + j*4] = arrayLocal[i*4 + j];
        }
    }
}

__global__ void DecFirstStageOverlapFilter(int* image, int numRows, int numCols)
{
    int arrayLocal_16[16], arrayLocal_4[4];
    int block_i = blockIdx.x, block_j = blockIdx.y, i, j;

    //numRows /= 4;
    //numCols /= 4;

    // 4x4 blocks
    for( i = 0; i < 4; i++)
        for( j = 0; j < 4; j++)
            arrayLocal_16[i*4+j] = image[((block_i*4 + i + 2)*4)*numCols + (block_j*4 + j + 2)*4];

    OverlapPostFilter4x4(arrayLocal_16);
    for( i = 0; i < 4; i++)
        for( j = 0; j < 4; j++)
            image[((block_i*4 + i + 2)*4)*numCols + (block_j*4 + j + 2)*4] = arrayLocal_16[i*4+j];
    //4x4 block end

    if(block_j == 0)
    {
        //left edge
        for(i = 0; i < 2; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[((block_i*4+j+2)*4)*numCols + i*4];
            OverlapPostFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[((block_i*4+j+2)*4)*numCols + i*4] = arrayLocal_4[j];
        }
        // right edge
        for(i = numCols/4-2; i < numCols/4; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[((block_i*4+j+2)*4)*numCols + i*4];
            OverlapPostFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[((block_i*4+j+2)*4)*numCols + i*4] = arrayLocal_4[j];
        }
    }


    if(block_i == 0)
    {
        //top edge
        for(i = 0; i < 2; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[(i*4)*numCols + (block_j*4+j+2)*4];
            OverlapPostFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[(i*4)*numCols + (block_j*4+j+2)*4] = arrayLocal_4[j];
        }
        //bottom edge
        for(i = numRows/4-2; i < numRows/4; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[(i*4)*numCols + (block_j*4+2+j)*4];
            OverlapPostFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[(i*4)*numCols + (block_j*4+j+2)*4] = arrayLocal_4[j];
        }
    }

    if(block_j == 0 && block_i == 0)
    {
        // top left
        arrayLocal_4[0] = image[(0)*numCols + 0], arrayLocal_4[1] = image[(0)*numCols + 4];
        arrayLocal_4[2] = image[(4)*numCols + 0], arrayLocal_4[3] = image[(4)*numCols + 4];
        OverlapPostFilter4(arrayLocal_4);
        image[(0)*numCols + 0] = arrayLocal_4[0], image[(0)*numCols + 4] = arrayLocal_4[1];
        image[(4)*numCols + 0] = arrayLocal_4[2], image[(4)*numCols + 4] = arrayLocal_4[3];

        // top right
        arrayLocal_4[0] = image[(0)*numCols + (numCols/4-2)*4], arrayLocal_4[1] = image[(0)*numCols + (numCols/4-1)*4];
        arrayLocal_4[2] = image[(4)*numCols + (numCols/4-2)*4], arrayLocal_4[3] = image[(4)*numCols + (numCols/4-1)*4];
        OverlapPostFilter4(arrayLocal_4);
        image[(0)*numCols + (numCols/4-2)*4] = arrayLocal_4[0], image[(0)*numCols + (numCols/4-1)*4] = arrayLocal_4[1];
        image[(4)*numCols + (numCols/4-2)*4] = arrayLocal_4[2], image[(4)*numCols + (numCols/4-1)*4] = arrayLocal_4[3];

        // bottom left
        arrayLocal_4[0] = image[((numRows/4-2)*4)*numCols + 0], arrayLocal_4[1] = image[((numRows/4-2)*4)*numCols + 4];
        arrayLocal_4[2] = image[((numRows/4-1)*4)*numCols + 0], arrayLocal_4[3] = image[((numRows/4-1)*4)*numCols + 4];
        OverlapPostFilter4(arrayLocal_4);
        image[((numRows/4-2)*4)*numCols + 0] = arrayLocal_4[0], image[((numRows/4-2)*4)*numCols + 4] = arrayLocal_4[1];
        image[((numRows/4-1)*4)*numCols + 0] = arrayLocal_4[2], image[((numRows/4-1)*4)*numCols + 4] = arrayLocal_4[3];

        // bottom right
        arrayLocal_4[0] = image[((numRows/4-2)*4)*numCols + (numCols/4-2)*4], arrayLocal_4[1] = image[((numRows/4-2)*4)*numCols + (numCols/4-1)*4];
        arrayLocal_4[2] = image[((numRows/4-1)*4)*numCols + (numCols/4-2)*4], arrayLocal_4[3] = image[((numRows/4-1)*4)*numCols + (numCols/4-1)*4];
        OverlapPostFilter4(arrayLocal_4);
        image[((numRows/4-2)*4)*numCols + (numCols/4-2)*4] = arrayLocal_4[0], image[((numRows/4-2)*4)*numCols + (numCols/4-1)*4] = arrayLocal_4[1];
        image[((numRows/4-1)*4)*numCols + (numCols/4-2)*4] = arrayLocal_4[2], image[((numRows/4-1)*4)*numCols + (numCols/4-1)*4] = arrayLocal_4[3];
    }
}

__global__ void DecSecondStageOverlapFilter(int *image, int numRows, int numCols)
{
    int arrayLocal_16[16], arrayLocal_4[4];
    int block_i = blockIdx.x, block_j = blockIdx.y, i, j;

    // 4x4 blocks
    for( i = 0; i < 4; i++)
    {
        for( j = 0; j < 4; j++)
            arrayLocal_16[i*4+j] = image[ (block_i*4 + 2 + i)*numCols + block_j*4 + 2 + j];
    }
    OverlapPostFilter4x4(arrayLocal_16);
    for( i = 0; i < 4; i++)
    {
        for( j = 0; j < 4; j++)
            image[(block_i*4 + 2 + i)*numCols + block_j*4 + 2 + j] = arrayLocal_16[i*4+j];
    }
    //4x4 block end

    if(block_j == 0)
    {
        //left edge
        for(i = 0; i < 2; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[(block_i*4 + 2+j)*numCols + i];
            OverlapPostFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[(block_i*4 + 2+j)*numCols + i] = arrayLocal_4[j];
        }
        // right edge
        for(i = numCols-2; i < numCols; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[(block_i*4 + 2 + j)*numCols + i];
            OverlapPostFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[(block_i*4 + 2+j)*numCols + i] = arrayLocal_4[j];
        }
    }


    if(block_i == 0)
    {
        //top edge
        for(i = 0; i < 2; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[(i)*numCols + block_j*4 + 2+j];
            OverlapPostFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[(i)*numCols + block_j*4 + 2+j] = arrayLocal_4[j];
        }
        //bottom edge
        for(i = numRows-2; i < numRows; i++)
        {
            for(j = 0; j < 4; j++)
                arrayLocal_4[j] = image[(i)*numCols + block_j*4 + 2 + j];
            OverlapPostFilter4(arrayLocal_4);
            for(j = 0; j < 4; j++)
                image[(i)*numCols + block_j*4 + 2 + j] = arrayLocal_4[j];
        }
    }

    if(block_i == 0 && block_j == 0)
    {
        // top left
        arrayLocal_4[0] = image[(0)*numCols + 0], arrayLocal_4[1] = image[(0)*numCols + 1];
        arrayLocal_4[2] = image[(1)*numCols + 0], arrayLocal_4[3] = image[(1)*numCols + 1];
        OverlapPostFilter4(arrayLocal_4);
        image[(0)*numCols + 0] = arrayLocal_4[0], image[(0)*numCols + 1] = arrayLocal_4[1];
        image[(1)*numCols + 0] = arrayLocal_4[2], image[(1)*numCols + 1] = arrayLocal_4[3];

        // top right
        arrayLocal_4[0] = image[(0)*numCols + numCols-2], arrayLocal_4[1] = image[(0)*numCols + numCols-1];
        arrayLocal_4[2] = image[(1)*numCols + numCols-2], arrayLocal_4[3] = image[(1)*numCols + numCols-1];
        OverlapPostFilter4(arrayLocal_4);
        image[(0)*numCols + numCols-2] = arrayLocal_4[0], image[(0)*numCols + numCols-1] = arrayLocal_4[1];
        image[(1)*numCols + numCols-2] = arrayLocal_4[2], image[(1)*numCols + numCols-1] = arrayLocal_4[3];

        // bottom left
        arrayLocal_4[0] = image[(numRows-2)*numCols + 0], arrayLocal_4[1] = image[(numRows-2)*numCols + 1];
        arrayLocal_4[2] = image[(numRows-1)*numCols + 0], arrayLocal_4[3] = image[(numRows-1)*numCols + 1];
        OverlapPostFilter4(arrayLocal_4);
        image[(numRows-2)*numCols + 0] = arrayLocal_4[0], image[(numRows-2)*numCols + 1] = arrayLocal_4[1];
        image[(numRows-1)*numCols + 0] = arrayLocal_4[2], image[(numRows-1)*numCols + 1] = arrayLocal_4[3];

        // bottom right
        arrayLocal_4[0] = image[(numRows-2)*numCols + numCols-2], arrayLocal_4[1] = image[(numRows-2)*numCols + numCols-1];
        arrayLocal_4[2] = image[(numRows-1)*numCols + numCols-2], arrayLocal_4[3] = image[(numRows-1)*numCols + numCols-1];
        OverlapPostFilter4(arrayLocal_4);
        image[(numRows-2)*numCols + numCols-2] = arrayLocal_4[0], image[(numRows-2)*numCols + numCols-1] = arrayLocal_4[1];
        image[(numRows-1)*numCols + numCols-2] = arrayLocal_4[2], image[(numRows-1)*numCols + numCols-1] = arrayLocal_4[3];
    }
}


int main()
{
    // read image in host
    int imageWidth = 112, imageHeight = 128;
    //scanf("%d %d", &imageHeight, &imageWidth);
    int image[128][112]; // = (int**) malloc(imageHeight * sizeof(int*) );
    int i, j;
    for(i = 0; i < imageHeight; i++){
        //image[i] = (int*) malloc(imageWidth * sizeof(int) );

        for(j = 0; j < imageWidth; j++)
            scanf( "%d", &image[i][j]);
    }

    // allocate & copy image memory in device
    int *imageDevice;
    int size = imageWidth * imageHeight * sizeof(int);
    hipMalloc((void**) &imageDevice, size );
    hipMemcpy(imageDevice, image, size, hipMemcpyHostToDevice);

    /* kernel invocation start */
    dim3 DimGrid(imageHeight/16, imageWidth/16);
    dim3 DimBlock(4, 4);
    dim3 DimGrid2(imageHeight/4-1, imageWidth/4-1);
    dim3 DimGrid3(imageHeight/16-1, imageWidth/16-1);
    // second stage Post-filtering
    DecSecondStagePostFiltering<<< DimGrid, 1>>>(imageDevice, imageHeight, imageWidth);
    // first stage frequency transform
    DecFirstStageOverlapFilter<<< DimGrid3, 1>>>(imageDevice, imageHeight, imageWidth);
    // first stage Post-filtering
    DecFirstStagePostFiltering<<< DimGrid, DimBlock>>>(imageDevice, imageHeight, imageWidth);
    // second stage frequency transform
    DecSecondStageOverlapFilter<<< DimGrid2, 1>>>(imageDevice, imageHeight, imageWidth);

    /* kernel function invocation end */

    // copy from device to host
    hipMemcpy(image, imageDevice, size, hipMemcpyDeviceToHost);

    //free device memory
    hipFree(imageDevice);

    //store processed image in file
    for( i = 0; i < imageHeight; i++)
    {
        for( j = 0; j < imageWidth; j++)
            fprintf(out, "%d ", image[i][j] );
        fprintf(out, "\n");
    }

    return 0;
}
